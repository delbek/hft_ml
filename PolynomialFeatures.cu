#include "hip/hip_runtime.h"
#include "PolynomialFeatures.h"
#include <cmath>

PolynomialFeatures::PolynomialFeatures(int trainingSize, int numFeatures, double* trainingData, int degree)
:
    m_TrainingSize(trainingSize),
    m_NumFeatures(numFeatures),
    m_TrainingData(trainingData),
    m_Degree(degree)
{}

double* PolynomialFeatures::transform()
{
    double* transformedData = new double[m_TrainingSize * m_NumFeatures * m_Degree];
    #pragma omp parallel for collapse(3) schedule(static) num_threads(NUM_THREADS)
    for (int i = 0; i < m_TrainingSize; ++i)
    {
        for (int j = 0; j < m_NumFeatures; ++j)
        {
            for (int k = 1; k <= m_Degree; ++k)
            {
                transformedData[i * m_NumFeatures * m_Degree + j * m_Degree + k] = pow(m_TrainingData[i * m_NumFeatures + j], k);
            }
        }
    }
    return transformedData;
}
