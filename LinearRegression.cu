#include "hip/hip_runtime.h"
#include "LinearRegression.h"
#include <random>
#include <omp.h>
#include <iostream>

LinearRegression::LinearRegression(int trainingSize, int numFeatures, double* trainingData, double* trainingLabels, 
                                double convergenceThreshold, double learningRate, int maxIterations)
:
    m_ConvergenceThreshold(convergenceThreshold), 
    m_LearningRate(learningRate),
    m_TrainingSize(trainingSize),
    m_NumFeatures(numFeatures),
    m_TrainingData(trainingData),
    m_TrainingLabels(trainingLabels),
    m_MaxIterations(maxIterations)
{
    m_Theta = new double[m_NumFeatures + 1];
    m_LastPredictions = new double[m_TrainingSize];
}

LinearRegression::~LinearRegression()
{
    delete[] m_Theta;
    delete[] m_LastPredictions;
}

void LinearRegression::initializeTheta()
{
    for (int j = 0; j < m_NumFeatures; ++j)
    {
        m_Theta[j] = 0;
    }
    m_Theta[m_NumFeatures] = 0;
}

void LinearRegression::fit()
{
    int iteration = 0;
    initializeTheta();
    double error = mse();
    std::cout << "Initial MSE: " << error << std::endl;
    while (error > m_ConvergenceThreshold && iteration < m_MaxIterations)
    {
        updateTheta();
        error = mse();
        ++iteration;
    }
    std::cout << "Final MSE: " << error << std::endl;
}

double LinearRegression::predict(double* x)
{
    double prediction = 0;
    for (int j = 0; j < m_NumFeatures; ++j)
    {
        prediction += m_Theta[j] * x[j];
    }
    prediction += m_Theta[m_NumFeatures];
    return prediction;
}

void LinearRegression::updateTheta()
{
    double* gradient = computeGradient();
    for (int j = 0; j < m_NumFeatures; ++j)
    {
        m_Theta[j] -= m_LearningRate * gradient[j];
    }
    m_Theta[m_NumFeatures] -= m_LearningRate * gradient[m_NumFeatures];
    delete[] gradient;
}

double LinearRegression::mse()
{
    double error = 0;
    #pragma omp parallel for reduction(+:error) schedule(static) num_threads(NUM_THREADS)
    for (int i = 0; i < m_TrainingSize; ++i)
    {
        double prediction = predict(m_TrainingData + i * m_NumFeatures);
        error += (prediction - m_TrainingLabels[i]) * (prediction - m_TrainingLabels[i]);
        m_LastPredictions[i] = prediction;
    }
    return error / m_TrainingSize;
}

double* LinearRegression::computeGradient()
{
    double* gradient = new double[m_NumFeatures + 1];
    std::fill(gradient, gradient + m_NumFeatures + 1, 0);
    
    int chunkSize = std::ceil(static_cast<double>(m_TrainingSize) / static_cast<double>(NUM_THREADS));

    #pragma omp parallel num_threads(NUM_THREADS)
    {
        int id = omp_get_thread_num();
        int myStart = id * chunkSize;
        int myEnd = std::min(myStart + chunkSize, m_TrainingSize);
        double* myGradient = new double[m_NumFeatures + 1];
        std::fill(myGradient, myGradient + m_NumFeatures + 1, 0);

        for (int i = myStart; i < myEnd; ++i)
        {
            double difference = m_LastPredictions[i] - m_TrainingLabels[i];
            for (int j = 0; j < m_NumFeatures; ++j)
            {
                myGradient[j] += difference * m_TrainingData[i * m_NumFeatures + j];
            }
            myGradient[m_NumFeatures] += difference;
        }
        for (int j = 0; j < m_NumFeatures; ++j)
        {
            #pragma omp atomic update
            gradient[j] += myGradient[j];
        }
        #pragma omp atomic update
        gradient[m_NumFeatures] += myGradient[m_NumFeatures];
        delete[] myGradient;
    }

    for (int j = 0; j < m_NumFeatures; ++j)
    {
        gradient[j] *= (double(2) / m_TrainingSize);
    }
    gradient[m_NumFeatures] *= (double(2) / m_TrainingSize);
 
    return gradient;
}
